#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "cuda_tool.h"
#include <hipDNN.h>

#define USE_CPU 1
const int N = 4096;  // matrix size
const int channel = 3;  // 通道数
const int K = 3;    // kernel size
const int convOutSize = N - K + 1;

__device__ float kernel_dev[K * K * channel];
// __constant__ float kernel_dev[K * K * channel];

void CPU_Conv(float *input, float *output, float *kernel) {
    for (int row = 0; row < convOutSize; row ++) {
        for (int col = 0; col < convOutSize; col ++) {
            float sum = 0.f;
            for (int i = 0; i < K; i ++) {
                for (int j = 0; j < K; j ++) {
                    int curCol = col + j;
                    int curRow = row + i;
                    // 3通道
                    sum += kernel[i * K + j] * input[curRow * N + curCol] + kernel[i * K + j + K * K] * input[curRow * N + curCol + N * N] + kernel[i * K + j + 2 * K * K] * input[curRow * N + curCol + 2 * N * N];
                }
            }
            output[row * convOutSize + col] = sum;
        }
    }
}

__global__ void Conv2DKernel(float *input, float *output) {
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    if (col >= convOutSize || row >= convOutSize) {
        return;
    }
    int curCol = 0, curRow = 0;
    float sum = 0.f;
    for (int i = 0; i < K; i ++) {
        for (int j = 0; j < K; j ++) {
            curCol = col + j;
            curRow = row + i;
            // 3 通道
            sum += kernel_dev[i * K + j] * input[curRow * N + curCol] + kernel_dev[i * K + j + K * K] * input[curRow * N + curCol + N * N] + kernel_dev[i * K + j + 2 * K * K] * input[curRow * N + curCol + 2 * N * N];
        }
    }
    output[row * convOutSize + col] = sum;
}

__device__ void matrix_mul(float *col_vec, float *res_dev) {
    const int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    const int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    if (nRow >= convOutSize || nCol >= convOutSize) return;

    const int col_vec_row = convOutSize * convOutSize;
    float c_sum = 0.f;
    for (int i = 0; i < K * K; i ++) {
        c_sum += kernel_dev[i] * col_vec[i * col_vec_row + nRow * convOutSize + nCol];
    }

    res_dev[nRow * convOutSize + nCol] = c_sum;
}

__global__ void im2col(float *input, float *col_vec, float *res_dev) {
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    if (col >= convOutSize || row >= convOutSize) {
        return;
    }
    int curCol = 0, curRow = 0;

    const int col_vec_row = convOutSize * convOutSize;
    // const int col_vec_col = K * K * 3;

    for (int i = 0; i < K; i ++) {
        for (int j = 0; j < K; j ++) {
            curRow = row + i;
            curCol = col + j;
            // 1. input
            // 2. col_vec: i * K + j 行
            //             row * convOutSize + col 列 
            col_vec[(i * K + j) * col_vec_row + row * convOutSize + col] = input[curRow * N + curCol];
            col_vec[(i * K + j) * col_vec_row + row * convOutSize + col + col_vec_row * K * K] = input[curRow * N + curCol + N * N];
            col_vec[(i * K + j) * col_vec_row + row * convOutSize + col + 2 * col_vec_row * K * K] = input[curRow * N + curCol + N * N * 2];
        }
    }
    __syncthreads();
    matrix_mul(col_vec, res_dev);
}

int main() {
    int imgSize = N * N * channel;

    float *matrix_host, *conved_m_host, *kernel_host, *cpu_conved_res;
    float *matrix_dev, *conved_m_dev, *col_vec_dev;

    int matrix_bytes = imgSize * sizeof(float);
    int conved_matrix_bytes = convOutSize * convOutSize * sizeof(float);
    int kernel_bytes = K * K * channel * sizeof(float);

    // 分配并初始化 host(CPU) 内存
    hipHostMalloc((void **) &matrix_host, matrix_bytes);
    hipHostMalloc((void **) &conved_m_host, conved_matrix_bytes);
    hipHostMalloc((void **) &cpu_conved_res, conved_matrix_bytes);
    hipHostMalloc((void **) &kernel_host, kernel_bytes);

    initialData(matrix_host, imgSize);
    initialData(kernel_host, K * K * channel);

    if (USE_CPU) {
        // cpu compute
        double iStart_cpu=cpuSecond();
        CPU_Conv(matrix_host, cpu_conved_res, kernel_host);
        double iElaps_cpu=cpuSecond()-iStart_cpu;
        printf("CPU \t\t\tExecution Time elapsed %f sec\n", iElaps_cpu);
    }

    // 分配device(GPU)上的内存
    hipMalloc((void **) &matrix_dev, matrix_bytes);
    hipMalloc((void **) &conved_m_dev, conved_matrix_bytes);
    // 分配拼接列向量矩阵的内存
    hipMalloc((void **) &col_vec_dev, K * K * 3 * convOutSize * convOutSize);

    hipMemcpy(matrix_dev, matrix_host, matrix_bytes, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(kernel_dev), kernel_host, kernel_bytes);

    dim3 block(K, K);
    dim3 grid((N + K - 1) / K, (N + K - 1) / K);

    double iStart_cuda = cpuSecond();
    Conv2DKernel<<<grid, block>>>(matrix_dev, conved_m_dev);
    double iElaps_cuda=cpuSecond()-iStart_cuda;
    printf("CUDA \t\t\tExecution Time elapsed %f sec\n", iElaps_cuda);
    hipMemcpy(conved_m_host, conved_m_dev, conved_matrix_bytes, hipMemcpyDeviceToHost);
    checkResult(cpu_conved_res, conved_m_host, convOutSize);

    double iStart_cuda_im2col = cpuSecond();
    im2col<<<grid, block>>>(matrix_dev, col_vec_dev, conved_m_dev);
    double iElaps_cuda_im2col=cpuSecond()-iStart_cuda_im2col;
    printf("CUDA_im2col: \t\tExecution Time elapsed %f sec\n", iElaps_cuda_im2col);

    hipMemcpy(conved_m_host, conved_m_dev, conved_matrix_bytes, hipMemcpyDeviceToHost);
    checkResult(cpu_conved_res, conved_m_host, convOutSize);

    // cudnn
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t in_desc;
    hipdnnCreateTensorDescriptor(&in_desc);
    hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, channel, N, N);

    hipdnnFilterDescriptor_t filt_desc;
    hipdnnCreateFilterDescriptor(&filt_desc);
    hipdnnSetFilter4dDescriptor(filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, channel, K, K);

    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor(conv_desc, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);

    int out_n, out_c, out_h, out_w;
    hipdnnGetConvolution2dForwardOutputDim(conv_desc, in_desc, filt_desc, &out_n, &out_c, &out_h, &out_w);

    float* out_data;
    hipMalloc(&out_data, out_n * out_c * out_h * out_w * sizeof(float));

    hipdnnTensorDescriptor_t out_desc;
    hipdnnCreateTensorDescriptor(&out_desc);
    hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w);

    size_t ws_size;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn, in_desc, filt_desc, conv_desc, out_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, &ws_size);

    float* ws_data;
    hipMalloc(&ws_data, ws_size);

    float alpha = 1.0f;
    float beta = 0.0f;

    double iStart_cudnn = cpuSecond();
    hipdnnConvolutionForward(
        cudnn,
        &alpha,
        in_desc, matrix_dev,
        filt_desc, kernel_host, // 修改这里，将kernel_host传给cuDNN
        conv_desc,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
        ws_data, ws_size,
        &beta,
        out_desc, out_data
    );
    double iElaps_cudnn = cpuSecond() - iStart_cudnn;
    printf("cuDNN \t\t\tExecution Time elapsed %f sec\n", iElaps_cudnn);

    float* host_out_data = new float[out_n * out_c * out_h * out_w];
    hipMemcpy(host_out_data, out_data, out_n * out_c * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);
    checkResult(cpu_conved_res, conved_m_host, convOutSize);

    hipFree(ws_data);
    hipFree(out_data);
    hipdnnDestroyTensorDescriptor(out_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroyFilterDescriptor(filt_desc);
    hipdnnDestroyTensorDescriptor(in_desc);
    hipdnnDestroy(cudnn);

    delete[] host_out_data;


    hipHostFree(matrix_host);
    hipHostFree(conved_m_host);
    hipHostFree(cpu_conved_res);
    hipHostFree(kernel_host);
    hipFree(matrix_dev);
    hipFree(conved_m_host);

    return 0;
}

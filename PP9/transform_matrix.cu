#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_tool.h"
#define BDIMX 8
#define BDIMY 8
#define IPAD 2
//cpu transform
void transformMatrix2D_CPU(float * in,float * out,int nx,int ny) {
  for(int j=0;j<ny;j++)
  {
    for(int i=0;i<nx;i++)
    {
      out[i*nx+j]=in[j*nx+i];
    }
  }
}

__global__ void transformNaiveRow(float * in,float * out,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx_row=ix+iy*nx;
    int idx_col=ix*ny+iy;
    if (ix<nx && iy<ny)
    {
      out[idx_col]=in[idx_row];
    }
}


//----------------------shared memory---------------------------
__global__ void transformSmem(float * in,float* out,int nx,int ny) {
	__shared__ float tile[BDIMY][BDIMX];
	unsigned int ix,iy,transform_in_idx,transform_out_idx;
	ix=threadIdx.x+blockDim.x*blockIdx.x;       // 全局的，没转置的ix
    iy=threadIdx.y+blockDim.y*blockIdx.y;       // 全局的，没转置的iy
	transform_in_idx=iy*nx+ix;                  // 全局的 in 数组的下标

	unsigned int bidx,irow,icol;                
	bidx=threadIdx.y*blockDim.x+threadIdx.x;    // bidx表示block idx也就是在这个块中的线程的坐标的线性位置
	irow=bidx/blockDim.y;                       // block转置后，的irow
	icol=bidx%blockDim.y;                       // block转置后，的icol


	ix=blockIdx.y*blockDim.y+icol;
	iy=blockIdx.x*blockDim.x+irow;


	transform_out_idx=iy*ny+ix;

	if(ix<nx&& iy<ny)
	{
		tile[threadIdx.y][threadIdx.x]=in[transform_in_idx];
		__syncthreads();
		out[transform_out_idx]=tile[icol][irow];

	}

}

__global__ void transformSmemPad(float * in,float* out,int nx,int ny) {
	__shared__ float tile[BDIMY][BDIMX+IPAD];
	unsigned int ix,iy,transform_in_idx,transform_out_idx;
	ix=threadIdx.x+blockDim.x*blockIdx.x;
    iy=threadIdx.y+blockDim.y*blockIdx.y;
	transform_in_idx=iy*nx+ix;

	unsigned int bidx,irow,icol;
	bidx=threadIdx.y*blockDim.x+threadIdx.x;
	irow=bidx/blockDim.y;
	icol=bidx%blockDim.y;


	ix=blockIdx.y*blockDim.y+icol;
	iy=blockIdx.x*blockDim.x+irow;


	transform_out_idx=iy*ny+ix;

	if(ix<nx&& iy<ny)
	{
		tile[threadIdx.y][threadIdx.x]=in[transform_in_idx];
		__syncthreads();
		out[transform_out_idx]=tile[icol][irow];

	}

}

int main(int argc,char** argv)
{
    initDevice(0);
    int nx=1<<12;
    int ny=1<<12;
    int dimx=BDIMX;
    int dimy=BDIMY;
    int nxy=nx*ny;
    int nBytes=nxy*sizeof(float);
    int transform_kernel=0;
    if(argc==2)
        transform_kernel=atoi(argv[1]);
    if(argc>=4)
    {
        transform_kernel=atoi(argv[1]);
        dimx=atoi(argv[2]);
        dimy=atoi(argv[3]);
    }

    //Malloc
    float* A_host=(float*)malloc(nBytes);
    float* B_host_cpu=(float*)malloc(nBytes);
    float* B_host=(float*)malloc(nBytes);
    initialData(A_host,nxy);

    //hipMalloc
    float *A_dev=NULL;
    float *B_dev=NULL;
    CHECK(hipMalloc((void**)&A_dev,nBytes));
    CHECK(hipMalloc((void**)&B_dev,nBytes));

    CHECK(hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice));
    CHECK(hipMemset(B_dev,0,nBytes));

    // cpu compute
    double iStart=cpuSecond();
    transformMatrix2D_CPU(A_host,B_host_cpu,nx,ny);
    double iElaps=cpuSecond()-iStart;
    printf("CPU Execution Time elapsed %f sec\n",iElaps);

    // 2d block and 2d grid
    dim3 block(dimx,dimy);
    dim3 grid((nx-1)/block.x+1,(ny-1)/block.y+1);
    dim3 block_1(dimx,dimy);
    dim3 grid_1((nx-1)/(block_1.x*2)+1,(ny-1)/block_1.y+1);

    CHECK(hipDeviceSynchronize());
    iStart=cpuSecond();
    switch(transform_kernel)
    {
    case 0:
            transformNaiveRow<<<grid,block>>>(A_dev,B_dev,nx,ny);
            printf("transformNaiveRow ");
            break;
    case 1:
            transformSmem<<<grid,block>>>(A_dev,B_dev,nx,ny);
            printf("transformSmem ");
            break;
    case 2:
            transformSmemPad<<<grid,block>>>(A_dev,B_dev,nx,ny);
            printf("transformSmemPad ");
            break;
    default:
        break;
    }
    CHECK(hipDeviceSynchronize());
    iElaps=cpuSecond()-iStart;
    printf(" Time elapsed %f sec\n",iElaps);
    CHECK(hipMemcpy(B_host,B_dev,nBytes,hipMemcpyDeviceToHost));
    checkResult(B_host,B_host_cpu,nxy);

    hipFree(A_dev);
    hipFree(B_dev);
    free(A_host);
    free(B_host);
    free(B_host_cpu);
    hipDeviceReset();
    return 0;
}
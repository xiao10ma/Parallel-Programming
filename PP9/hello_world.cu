
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world(void)
{
    printf("Hello World from Thread(%d, %d) in Block %d!\n", threadIdx.x, threadIdx.y, blockIdx.x);
}

int main(int argc, char **argv)
{
    const int N = 15, M = 5, K = 5;

    dim3 block(M, K);
    dim3 grid(N);

    printf("Hello World from the host!\n");
    hello_world<<<grid, block>>>();
    hipDeviceReset(); // if no this line ,it can not output hello world from gpu
    return 0;
}

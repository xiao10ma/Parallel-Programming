#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_tool.h"

#define USE_CPU 1
const int N = 1024;
const int block_size = 16;

void mm_cpu(float *A_host, float *B_host, float *C_host) {
    for (int i = 0; i < N; i ++) {
        for (int k = 0; k < N; k ++) {
            for (int j = 0; j < N; j ++) {
                C_host[i * N + j] += A_host[i * N + k] * B_host[k * N + j];
            }
        }
    }
}

__global__ void mm_cuda(float *A_dev, float *B_dev, float *C_dev) {
    int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    float c_sum = 0.f;

    for (int i = 0; i < N; i ++) {
        c_sum += A_dev[nRow * N + i] * B_dev[i * N + nCol];
    }
    C_dev[nRow * N + nCol] = c_sum;
}

__global__ void mm_shared_mem(float *A_dev, float *B_dev, float *C_dev) {
    int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;

    float c_sum = 0.f;

    __shared__ float A_tile[block_size][block_size];
    __shared__ float B_tile[block_size][block_size];

    int nIter = (N + block_size - 1) / block_size;      // 将矩阵一个维度拆成 nIter 块
    for (int i = 0; i < nIter; i ++) {
        A_tile[threadIdx.y][threadIdx.x] = A_dev[nRow * N + i * block_size + threadIdx.x];
        B_tile[threadIdx.y][threadIdx.x] = B_dev[(i * block_size + threadIdx.y) * N + nCol];
        // 同步block中不同warp
        __syncthreads();

        for (int iter = 0; iter < block_size; iter ++) {
            c_sum += A_tile[threadIdx.y][iter] * B_tile[iter][threadIdx.x];
        }
        // 同步block中不同warp
        __syncthreads();
    }
    C_dev[nRow * N + nCol] = c_sum;
}

int main() 
{
    float *A_host, *B_host, *C_host, *C_dev2host, *C_dev2host_shared;
    float *A_dev, *B_dev, *C_dev, *C_dev_shared;
    int nxy = N * N;    // matrix size
    int nBytes = sizeof(float) * N * N;

    // 分配host(CPU)上的内存
    hipHostMalloc((void **) &A_host, nBytes);
    hipHostMalloc((void **) &B_host, nBytes);
    hipHostMalloc((void **) &C_host, nBytes);
    hipHostMalloc((void **) &C_dev2host, nBytes);
    hipHostMalloc((void **) &C_dev2host_shared, nBytes);

    initialData(A_host, nxy);
    initialData(B_host, nxy);

    if (USE_CPU) {
        // cpu compute
        double iStart_cpu=cpuSecond();
        mm_cpu(A_host, B_host, C_host);
        double iElaps_cpu=cpuSecond()-iStart_cpu;
        printf("CPU \t\t\tExecution Time elapsed %f sec\n", iElaps_cpu);
    }

    // 分配device(GPU)上的内存
    hipMalloc((void **) &A_dev, nBytes);
    hipMalloc((void **) &B_dev, nBytes);
    hipMalloc((void **) &C_dev, nBytes);
    hipMalloc((void **) &C_dev_shared, nBytes);

    hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(B_dev, B_host, nBytes, hipMemcpyHostToDevice);

    dim3 block(block_size, block_size);
    dim3 grid((N + block_size - 1) / block_size, (N + block_size - 1) / block_size);

    double iStart_cuda=cpuSecond();
    mm_cuda<<<grid, block>>>(A_dev, B_dev, C_dev);
    hipDeviceSynchronize();
    double iElaps_cuda=cpuSecond()-iStart_cuda;
    printf("CUDA \t\t\tExecution Time elapsed %f sec\n", iElaps_cuda);

    double iStart_shared=cpuSecond();
    mm_shared_mem<<<grid, block>>>(A_dev, B_dev, C_dev_shared);
    hipDeviceSynchronize();
    double iElaps_shared=cpuSecond()-iStart_shared;
    printf("CUDA(shared mem) \tExecution Time elapsed %f sec\n", iElaps_shared);
    printf("---------------------------------------------\n");
    hipDeviceSynchronize();
    hipMemcpy(C_dev2host, C_dev, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(C_dev2host_shared, C_dev_shared, nBytes, hipMemcpyDeviceToHost);
    printf("CUDA: \t\t\t");
    checkResult(C_host, C_dev2host, nxy);
    printf("CUDA(shared mem): \t");
    checkResult(C_host, C_dev2host_shared, nxy);

    hipHostFree(A_host);
    hipHostFree(B_host);
    hipHostFree(C_host);
    hipHostFree(C_dev2host);
    hipHostFree(C_dev2host_shared);
    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(C_dev);
    hipFree(C_dev_shared);
}
